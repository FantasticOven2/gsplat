#include "hip/hip_runtime.h"
#include "bindings.h"
#include "utils.cuh"
#include "helpers.cuh"
#include "types.cuh"
#include <hip/hip_cooperative_groups.h>
#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime.h>

namespace cg = cooperative_groups;

/****************************************************************************
 * Rasterization to Pixels Backward Pass 2DGS
 ****************************************************************************/
template <uint32_t COLOR_DIM, typename S>
__global__ void rasterize_to_pixels_bwd_2dgs_kernel(
    const uint32_t C, const uint32_t N, const uint32_t n_isects, const bool packed,
    // fwd inputs
    const vec2<S> *__restrict__ means2d,
    const S *__restrict__ ray_transformations,
    const S *__restrict__ colors,
    const vec3<S> *__restrict__ normals,
    const S *__restrict__ opacities,
    const S *__restrict__ backgrounds,
    const uint32_t image_width, const uint32_t image_height, const uint32_t tile_size,
    const uint32_t tile_width, const uint32_t tile_height,
    const int32_t *__restrict__ tile_offsets,
    const int32_t *__restrict__ flatten_ids,
    // fwd outputs
    const S *__restrict__ render_alphas,
    const int32_t *__restrict__ last_ids,
    // grad outputs
    const S *__restrict__ v_render_colors,
    const S *__restrict__ v_render_alphas,
    const S *__restrict__ v_render_normals,
    // grad inputs
    vec2<S> *__restrict__ v_means2d_abs,
    vec2<S> *__restrict__ v_means2d,
    S *__restrict__ v_ray_transformations,
    S *__restrict__ v_colors,
    S *__restrict__ v_opacities,
    S *__restrict__ v_normal3d
) {
    auto block = cg::this_thread_block();
    uint32_t camera_id = block.group_index().x;
    uint32_t tile_id = block.group_index().y * tile_width + block.group_index().z;
    uint32_t i = block.group_index().y * tile_size + block.thread_index().y;
    uint32_t j = block.group_index().z * tile_size + block.thread_index().x;

    tile_offsets += camera_id * tile_height * tile_width;
    render_alphas += camera_id * image_height * image_width;
    last_ids += camera_id * image_height * image_width;
    v_render_colors += camera_id * image_height * image_width * COLOR_DIM;
    v_render_alphas += camera_id * image_height * image_width;
    v_render_normals += camera_id * image_height * image_width * 3;
    if (backgrounds != nullptr) {
        backgrounds += camera_id * COLOR_DIM;
    }


    const S px = (S)j + 0.5f;
    const S py = (S)i + 0.5f;
    // clamp this value to the last pixel
    const int32_t pix_id = min(i * image_width + j, image_width * image_height - 1);

    // keep not rasterizing threads around for reading data
    bool inside = (i < image_height && j < image_width);

    // have all threads in tile process the same gaussians in batches
    // first collect gaussians between range.x and range.y in batches
    // which gaussians to look through in this tile
    int32_t range_start = tile_offsets[tile_id];
    int32_t range_end = 
        (camera_id == C - 1) && (tile_id == tile_width * tile_height - 1)
            ? n_isects
            : tile_offsets[tile_id + 1];
    const uint32_t block_size = block.size();
    const uint32_t num_batches = 
        (range_end - range_start + block_size - 1) / block_size;
    
    // __shared__ int32_t id_batch[MAX_BLOCK_SIZE];
    // __shared__ vec3<S> xy_opacity_batch[MAX_BLOCK_SIZE];
    // __shared__ vec3<S> u_transform_batch[MAX_BLOCK_SIZE];
    // __shared__ vec3<S> v_transform_batch[MAX_BLOCK_SIZE];
    // __shared__ vec3<S> w_transform_batch[MAX_BLOCK_SIZE];
    // __shared__ S rgbs_batch[MAX_BLOCK_SIZE * COLOR_DIM];
    // __shared__ S normals_batch[MAX_BLOCK_SIZE * 3];

    extern __shared__ int s[];
    int32_t *id_batch = (int32_t *)s; // [block_size]
    vec3<S> *xy_opacity_batch =
        reinterpret_cast<vec3<float> *>(&id_batch[block_size]); // [block_size]
    vec3<S> *u_transform_batch =
        reinterpret_cast<vec3<float> *>(&xy_opacity_batch[block_size]); // [block_size]
    vec3<S> *v_transform_batch = 
        reinterpret_cast<vec3<float> *>(&u_transform_batch[block_size]);
    vec3<S> *w_transform_batch = 
        reinterpret_cast<vec3<float> *>(&v_transform_batch[block_size]);
    S *rgbs_batch = (S *)&w_transform_batch[block_size]; // [block_size * COLOR_DIM]
    S *normals_batch = (S *)&rgbs_batch[block_size * COLOR_DIM]; // [block_size * 3]

    // this is the T AFTER the last gaussian in this pixel
    S T_final = 1.0f - render_alphas[pix_id];
    S T = T_final;
    // the contribution from gaussians behind the current one
    S buffer[COLOR_DIM] = {0.f};
    S buffer_normals[3] = {0.f};
    // index of last gaussian to contribute to this pixel
    const int32_t bin_final = inside ? last_ids[pix_id] : 0;

    // df/d_out for this pixel
    S v_render_c[COLOR_DIM];
    PRAGMA_UNROLL
    for (uint32_t k = 0; k < COLOR_DIM; ++k) {
        v_render_c[k] = v_render_colors[pix_id * COLOR_DIM + k];
    }
    const S v_render_a = v_render_alphas[pix_id];
    S v_render_n[3];
    PRAGMA_UNROLL
    for (uint32_t k = 0; k < 3; ++k) {
        v_render_n[k] = v_render_normals[pix_id * 3 + k];
    }



    // collect and process batches of gaussians
    // each thread loads one gaussian at a time before rasterizing
    const uint32_t tr = block.thread_rank();
    cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
    const int32_t warp_bin_final = cg::reduce(warp, bin_final, cg::greater<int>());
    for (uint32_t b = 0; b < num_batches; ++b) {
        // resync all threads before writing next batch of shared mem
        block.sync();

        // each thread fetch 1 gaussian from back to front 
        // 0 index will be furthest back in batch
        // index of gaussian to load
        // batch end is the index of the last gaussian in the batch
        // These values can be negative so must be int32 instead of uint32
        const int32_t batch_end = range_end - 1 - block_size * b;
        const int32_t batch_size = min(block_size, batch_end + 1 - range_start);
        const int32_t idx = batch_end - tr;
        if (idx >= range_start) {
            int32_t g = flatten_ids[idx]; // flatten index in [C * N] or [nnz]
            id_batch[tr] = g;
            const vec2<S> xy = means2d[g];
            const S opac = opacities[g];
            xy_opacity_batch[tr] = {xy.x, xy.y, opac};
            u_transform_batch[tr] = {ray_transformations[g * 9 + 0], ray_transformations[g * 9 + 1], ray_transformations[g * 9 + 2]};
            v_transform_batch[tr] = {ray_transformations[g * 9 + 3], ray_transformations[g * 9 + 4], ray_transformations[g * 9 + 5]};
            w_transform_batch[tr] = {ray_transformations[g * 9 + 6], ray_transformations[g * 9 + 7], ray_transformations[g * 9 + 8]};

            const vec3<S> normal = normals[g];
            normals_batch[tr * 3] = normal.x;
            normals_batch[tr * 3 + 1] = normal.y;
            normals_batch[tr * 3 + 2] = normal.z;

            PRAGMA_UNROLL
            for (uint32_t k = 0; k < COLOR_DIM; ++k) {
                rgbs_batch[tr * COLOR_DIM + k] = colors[g * COLOR_DIM + k];
            }

        }

        // wait for other threads to collect the gaussians in batch
        block.sync();
        // process gaussians in the current batch for this pixel
        // 0 index is the furthest back gaussian in the batch
        for (uint32_t t = max(0, batch_end - warp_bin_final); t < batch_size; ++t) {
            bool valid = inside;
            if (batch_end - t > bin_final) {
                valid = 0;
            }
            S alpha;
            S opac;
            S vis; 
            S gauss_weight_3d;
            S gauss_weight_2d;
            S gauss_weight;
            vec2<S> s;
            vec2<S> d;
            vec3<S> h_u;
            vec3<S> h_v;
            vec3<S> intersect;
            vec3<S> w_transform;
            if (valid) { 
                vec3<S> xy_opac = xy_opacity_batch[t];
                opac = xy_opac.z;
                const vec3<S> u_transform = u_transform_batch[t];
                const vec3<S> v_transform = v_transform_batch[t];
                w_transform = w_transform_batch[t];

                h_u = px * w_transform - u_transform;
                h_v = py * w_transform - v_transform;
                

                // cross product of two planes is a line
                intersect = cross_product(h_u, h_v);

                // No intersection
                if (intersect.z == 0.0) valid = false;
                s = {intersect.x / intersect.z, intersect.y / intersect.z};

                gauss_weight_3d = s.x * s.x + s.y * s.y;
                d = {xy_opac.x - px, xy_opac.y - py};
                gauss_weight_2d = FilterInvSquare * (d.x * d.x + d.y * d.y);

                gauss_weight = min(gauss_weight_3d, gauss_weight_2d);
                const S sigma = 0.5f * gauss_weight;
                vis = __expf(-sigma);
                alpha = min(0.999f, opac * vis);
                if (sigma < 0.f || alpha < 1.f / 255.f) {
                    valid = false;
                }
            }


            // if all threads are inactive in this warp, skip this loop
            if (!warp.any(valid)) {
                continue;
            }
            S v_rgb_local[COLOR_DIM] = {0.f};
            vec3<S> v_u_transform_local = {0.f, 0.f, 0.f};
            vec3<S> v_v_transform_local = {0.f, 0.f, 0.f};
            vec3<S> v_w_transform_local = {0.f, 0.f, 0.f};
            vec2<S> v_xy_local = {0.f, 0.f};
            vec2<S> v_xy_abs_local = {0.f, 0.f};
            S v_opacity_local = 0.f;
            S v_normal_local[3] = {0.f};
            // initialize everything to 0, only set if the lane is valid
            if (valid) {
                // compute the current T for this gaussian
                S ra = 1.0f / (1.0f - alpha);
                T *= ra;
                // update v_rgb for this gaussian
                const S fac = alpha * T;
                PRAGMA_UNROLL
                for (uint32_t k = 0; k < COLOR_DIM; ++k) {
                    v_rgb_local[k] = fac * v_render_c[k];
                }
                // contribution from this pixel
                S v_alpha = 0.f;
                for (uint32_t k = 0; k < COLOR_DIM; ++k) {
                    v_alpha += (rgbs_batch[t * COLOR_DIM + k] * T - buffer[k] * ra) *
                                v_render_c[k];
                }

                // update v_normal for this gaussian
                // TODO (WZ): derive the computational graph to see if the gradient flow
                // is correct or not.
                PRAGMA_UNROLL
                for (uint32_t k = 0; k < 3; ++k) {
                    v_normal_local[k] = fac * v_render_n[k];
                }

                for (uint32_t k = 0; k < 3; ++k) {
                    v_alpha += (normals_batch[t * 3 + k] * T - buffer_normals[k] * ra) *
                                v_render_n[k];
                }

                v_alpha += T_final * ra * v_render_a;

                // contribution from background pixel
                if (backgrounds != nullptr) {
                    S accum = 0.f;
                    PRAGMA_UNROLL
                    for (uint32_t k = 0; k < COLOR_DIM; ++k) {
                        accum += backgrounds[k] * v_render_c[k];
                    }
                    v_alpha += -T_final * ra * accum;
                }


                //====== 2DGS ======//
                if (opac * vis <= 0.999f) {
                    const S v_G = opac * v_alpha;
                    S v_depth = 0.f;
                    if (gauss_weight_3d <= gauss_weight_2d) {
                        const vec2<S> v_s = {
                            v_G * -vis * s.x + v_depth * w_transform.x,
                            v_G * -vis * s.y + v_depth * w_transform.y
                        };
                        const vec3<S> v_z_w_transform = {s.x, s.y, 1.0};
                        const S v_sx_pz = v_s.x / intersect.z;
                        const S v_sy_pz = v_s.y / intersect.z;
                        const vec3<S> v_intersect = {v_sx_pz, v_sy_pz, -(v_sx_pz * s.x + v_sy_pz * s.y)};
                        
                        
                        const vec3<S> v_h_u = cross_product(h_v, v_intersect);
                        const vec3<S> v_h_v = cross_product(v_intersect, h_u);
                        
                        v_u_transform_local = {-v_h_u.x, -v_h_u.y, -v_h_u.z};
                        v_v_transform_local = {-v_h_v.x, -v_h_v.y, -v_h_v.z};
                        v_w_transform_local = {
                            px * v_h_u.x + py * v_h_v.x + v_depth * v_z_w_transform.x,
                            px * v_h_u.y + py * v_h_v.y + v_depth * v_z_w_transform.y,
                            px * v_h_u.z + py * v_h_v.z + v_depth * v_z_w_transform.z
                        };
                        
                    } else {
                        const S v_G_ddelx = -vis * FilterInvSquare * d.x;
                        const S v_G_ddely = -vis * FilterInvSquare * d.y;
                        v_xy_local = {v_G * v_G_ddelx, v_G * v_G_ddely};
                    }

                    v_opacity_local = vis * v_alpha;
                }

                PRAGMA_UNROLL
                for (uint32_t k = 0; k < COLOR_DIM; ++k) {
                    buffer[k] += rgbs_batch[t * COLOR_DIM + k] * fac;
                }

                PRAGMA_UNROLL
                for (uint32_t k = 0; k < 3; ++k) {
                    buffer_normals[k] += normals_batch[t * 3 + k] * fac;
                }
                
            }
            warpSum<COLOR_DIM, S>(v_rgb_local, warp);
            warpSum<3, S>(v_normal_local, warp);
            warpSum<decltype(warp), S>(v_xy_local, warp);
            if (v_means2d_abs != nullptr) {
                warpSum<decltype(warp), S>(v_xy_abs_local, warp);
            }
            warpSum<decltype(warp), S>(v_opacity_local, warp);
            warpSum<decltype(warp), S>(v_u_transform_local, warp);
            warpSum<decltype(warp), S>(v_v_transform_local, warp);
            warpSum<decltype(warp), S>(v_w_transform_local, warp);
            if (warp.thread_rank() == 0) {
                int32_t g = id_batch[t]; // flatten index in [C * N] or [nnz]
                S *v_rgb_ptr = (S *)(v_colors) + COLOR_DIM * g;
                PRAGMA_UNROLL
                for (uint32_t k = 0; k < COLOR_DIM; ++k) {
                    gpuAtomicAdd(v_rgb_ptr + k, v_rgb_local[k]);
                }

                //====== 2DGS ======//
                // if (gauss_weight_3d <= gauss_weight_2d) {
                S *v_ray_transformation_ptr = (S *)(v_ray_transformations) + 9 * g;
                gpuAtomicAdd(v_ray_transformation_ptr, v_u_transform_local.x);
                gpuAtomicAdd(v_ray_transformation_ptr + 1, v_u_transform_local.y);
                gpuAtomicAdd(v_ray_transformation_ptr + 2, v_u_transform_local.z);
                gpuAtomicAdd(v_ray_transformation_ptr + 3, v_v_transform_local.x);
                gpuAtomicAdd(v_ray_transformation_ptr + 4, v_v_transform_local.y);
                gpuAtomicAdd(v_ray_transformation_ptr + 5, v_v_transform_local.z);
                gpuAtomicAdd(v_ray_transformation_ptr + 6, v_w_transform_local.x);
                gpuAtomicAdd(v_ray_transformation_ptr + 7, v_w_transform_local.y);
                gpuAtomicAdd(v_ray_transformation_ptr + 8, v_w_transform_local.z);
                // } else {
                // printf("%.2f, %.2f \n", gauss_weight_3d, gauss_weight_2d);
                S *v_xy_ptr = (S *)(v_means2d) + 2 * g;
                gpuAtomicAdd(v_xy_ptr, v_xy_local.x);
                gpuAtomicAdd(v_xy_ptr + 1, v_xy_local.y);

                if (v_means2d_abs != nullptr) {
                    S *v_xy_abs_ptr = (S *)(v_means2d_abs) + 2 * g;
                    gpuAtomicAdd(v_xy_abs_ptr, v_xy_abs_local.x);
                    gpuAtomicAdd(v_xy_abs_ptr + 1, v_xy_abs_local.y);
                }
                // }


                S *v_normal_ptr = (S *)(v_normal3d) + 3 * g;
                PRAGMA_UNROLL
                for (uint32_t k = 0; k < 3; ++k) {
                    gpuAtomicAdd(v_normal_ptr + k, v_normal_local[k]);
                }
                
                gpuAtomicAdd(v_opacities + g, v_opacity_local);
            }
        }
    }
}

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor>
rasterize_to_pixels_bwd_2dgs_tensor(
    // Gaussian parameters
    const torch::Tensor &means2d,
    const torch::Tensor &ray_transformations,
    const torch::Tensor &colors,
    const torch::Tensor &opacities,
    const torch::Tensor &normals,
    const at::optional<torch::Tensor> &backgrounds,
    // image size
    const uint32_t image_width, const uint32_t image_height, const uint32_t tile_size,
    // intersection
    const torch::Tensor &tile_offsets,
    const torch::Tensor &flatten_ids,
    // forward outputs
    const torch::Tensor &render_alphas,
    const torch::Tensor &last_ids,
    // gradients of outputs
    const torch::Tensor &v_render_colors,
    const torch::Tensor &v_render_alphas,
    const torch::Tensor &v_render_normals,
    // options
    bool absgrad
) {
    DEVICE_GUARD(means2d);
    CHECK_INPUT(means2d);
    CHECK_INPUT(ray_transformations);
    CHECK_INPUT(colors);
    CHECK_INPUT(opacities);
    CHECK_INPUT(normals);
    CHECK_INPUT(tile_offsets);
    CHECK_INPUT(flatten_ids);
    CHECK_INPUT(render_alphas);
    CHECK_INPUT(last_ids);
    CHECK_INPUT(v_render_colors);
    CHECK_INPUT(v_render_alphas);
    CHECK_INPUT(v_render_normals);
    if (backgrounds.has_value()) {
        CHECK_INPUT(backgrounds.value());
    }

    bool packed = means2d.dim() == 2;

    uint32_t C = tile_offsets.size(0);         // number of cameras
    uint32_t N = packed ? 0 : means2d.size(1); // number of gaussians
    uint32_t n_isects = flatten_ids.size(0);
    uint32_t COLOR_DIM = colors.size(-1);
    uint32_t tile_height = tile_offsets.size(1);
    uint32_t tile_width = tile_offsets.size(2);

    // Each block covers a tile on the image. In total there are
    // C * tile_height * tile_width blocks.
    dim3 threads = {tile_size, tile_size, 1};
    dim3 blocks = {C, tile_height, tile_width};

    torch::Tensor v_means2d = torch::zeros_like(means2d);
    torch::Tensor v_ray_transformations = torch::zeros_like(ray_transformations);
    torch::Tensor v_colors = torch::zeros_like(colors);
    torch::Tensor v_opacities = torch::zeros_like(opacities);
    torch::Tensor v_means2d_abs;
    if (absgrad) {
        v_means2d_abs = torch::zeros_like(means2d);
    }
    torch::Tensor v_normals = torch::zeros_like(normals);
    if (n_isects) {
        const uint32_t shared_mem = tile_size * tile_size *
                                    (sizeof(int32_t) + sizeof(vec3<float>) +
                                     sizeof(vec3<float>) + +sizeof(vec3<float>) + 
                                     sizeof(vec3<float>) + sizeof(float) * COLOR_DIM
                                     +sizeof(float) * 3);
        at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();

        // if (hipFuncSetAttribute(reinterpret_cast<const void*>(rasterize_to_pixels_bwd_2dgs_kernel<COLOR_DIM), float>,
        //                          hipFuncAttributeMaxDynamicSharedMemorySize,
        //                          shared_mem) != hipSuccess) {
        //     AT_ERROR("Failed to set maximum shared memory size (requested ", shared_mem,
        //              " bytes), try lowering tile_size.");
        // }
        switch (COLOR_DIM) {
        case 1:
            rasterize_to_pixels_bwd_2dgs_kernel<1, float><<<blocks, threads, shared_mem, stream>>>(
                C, N, n_isects, packed, reinterpret_cast<vec2<float> *>(means2d.data_ptr<float>()),
                ray_transformations.data_ptr<float>(), colors.data_ptr<float>(),
                reinterpret_cast<vec3<float> *>(normals.data_ptr<float>()), opacities.data_ptr<float>(),
                backgrounds.has_value() ? backgrounds.value().data_ptr<float>()
                                        : nullptr,
                image_width, image_height, tile_size, tile_width, tile_height,
                tile_offsets.data_ptr<int32_t>(), flatten_ids.data_ptr<int32_t>(),
                render_alphas.data_ptr<float>(), 
                last_ids.data_ptr<int32_t>(),
                v_render_colors.data_ptr<float>(), v_render_alphas.data_ptr<float>(),
                v_render_normals.data_ptr<float>(),
                absgrad ? reinterpret_cast<vec2<float> *>(v_means2d_abs.data_ptr<float>()) : nullptr,
                reinterpret_cast<vec2<float> *>(v_means2d.data_ptr<float>()),
                v_ray_transformations.data_ptr<float>(), v_colors.data_ptr<float>(),
                v_opacities.data_ptr<float>(), v_normals.data_ptr<float>()
            );
            break;
        case 2:
            rasterize_to_pixels_bwd_2dgs_kernel<2, float><<<blocks, threads, shared_mem, stream>>>(
                C, N, n_isects, packed, reinterpret_cast<vec2<float> *>(means2d.data_ptr<float>()),
                ray_transformations.data_ptr<float>(), colors.data_ptr<float>(),
                reinterpret_cast<vec3<float> *>(normals.data_ptr<float>()), opacities.data_ptr<float>(),
                backgrounds.has_value() ? backgrounds.value().data_ptr<float>()
                                        : nullptr,
                image_width, image_height, tile_size, tile_width, tile_height,
                tile_offsets.data_ptr<int32_t>(), flatten_ids.data_ptr<int32_t>(),
                render_alphas.data_ptr<float>(), 
                last_ids.data_ptr<int32_t>(),
                v_render_colors.data_ptr<float>(), v_render_alphas.data_ptr<float>(),
                v_render_normals.data_ptr<float>(),
                absgrad ? reinterpret_cast<vec2<float> *>(v_means2d_abs.data_ptr<float>()) : nullptr,
                reinterpret_cast<vec2<float> *>(v_means2d.data_ptr<float>()),
                v_ray_transformations.data_ptr<float>(), v_colors.data_ptr<float>(),
                v_opacities.data_ptr<float>(), v_normals.data_ptr<float>()
            );
            break;
        case 3:
            rasterize_to_pixels_bwd_2dgs_kernel<3, float><<<blocks, threads, shared_mem, stream>>>(
                C, N, n_isects, packed, reinterpret_cast<vec2<float> *>(means2d.data_ptr<float>()),
                ray_transformations.data_ptr<float>(), colors.data_ptr<float>(),
                reinterpret_cast<vec3<float> *>(normals.data_ptr<float>()), opacities.data_ptr<float>(),
                backgrounds.has_value() ? backgrounds.value().data_ptr<float>()
                                        : nullptr,
                image_width, image_height, tile_size, tile_width, tile_height,
                tile_offsets.data_ptr<int32_t>(), flatten_ids.data_ptr<int32_t>(),
                render_alphas.data_ptr<float>(), 
                last_ids.data_ptr<int32_t>(),
                v_render_colors.data_ptr<float>(), v_render_alphas.data_ptr<float>(),
                v_render_normals.data_ptr<float>(),
                absgrad ? reinterpret_cast<vec2<float> *>(v_means2d_abs.data_ptr<float>()) : nullptr,
                reinterpret_cast<vec2<float> *>(v_means2d.data_ptr<float>()),
                v_ray_transformations.data_ptr<float>(), v_colors.data_ptr<float>(),
                v_opacities.data_ptr<float>(), v_normals.data_ptr<float>()
            );
            break;     
        case 4:
            rasterize_to_pixels_bwd_2dgs_kernel<4, float><<<blocks, threads, shared_mem, stream>>>(
                C, N, n_isects, packed, reinterpret_cast<vec2<float> *>(means2d.data_ptr<float>()),
                ray_transformations.data_ptr<float>(), colors.data_ptr<float>(),
                reinterpret_cast<vec3<float> *>(normals.data_ptr<float>()), opacities.data_ptr<float>(),
                backgrounds.has_value() ? backgrounds.value().data_ptr<float>()
                                        : nullptr,
                image_width, image_height, tile_size, tile_width, tile_height,
                tile_offsets.data_ptr<int32_t>(), flatten_ids.data_ptr<int32_t>(),
                render_alphas.data_ptr<float>(), 
                last_ids.data_ptr<int32_t>(),
                v_render_colors.data_ptr<float>(), v_render_alphas.data_ptr<float>(),
                v_render_normals.data_ptr<float>(),
                absgrad ? reinterpret_cast<vec2<float> *>(v_means2d_abs.data_ptr<float>()) : nullptr,
                reinterpret_cast<vec2<float> *>(v_means2d.data_ptr<float>()),
                v_ray_transformations.data_ptr<float>(), v_colors.data_ptr<float>(),
                v_opacities.data_ptr<float>(), v_normals.data_ptr<float>()
            );
            // CUDA_CHECK_ERROR;
            // CUDA_SAFE_CALL(hipStreamSynchronize(stream.stream()));
            break;
        case 8:
            rasterize_to_pixels_bwd_2dgs_kernel<8, float><<<blocks, threads, shared_mem, stream>>>(
                C, N, n_isects, packed, reinterpret_cast<vec2<float> *>(means2d.data_ptr<float>()),
                ray_transformations.data_ptr<float>(), colors.data_ptr<float>(),
                reinterpret_cast<vec3<float> *>(normals.data_ptr<float>()), opacities.data_ptr<float>(),
                backgrounds.has_value() ? backgrounds.value().data_ptr<float>()
                                        : nullptr,
                image_width, image_height, tile_size, tile_width, tile_height,
                tile_offsets.data_ptr<int32_t>(), flatten_ids.data_ptr<int32_t>(),
                render_alphas.data_ptr<float>(), 
                last_ids.data_ptr<int32_t>(),
                v_render_colors.data_ptr<float>(), v_render_alphas.data_ptr<float>(),
                v_render_normals.data_ptr<float>(),
                absgrad ? reinterpret_cast<vec2<float> *>(v_means2d_abs.data_ptr<float>()) : nullptr,
                reinterpret_cast<vec2<float> *>(v_means2d.data_ptr<float>()),
                v_ray_transformations.data_ptr<float>(), v_colors.data_ptr<float>(),
                v_opacities.data_ptr<float>(), v_normals.data_ptr<float>()
            );
            break;
        case 16:
            rasterize_to_pixels_bwd_2dgs_kernel<16, float><<<blocks, threads, shared_mem, stream>>>(
                C, N, n_isects, packed, reinterpret_cast<vec2<float> *>(means2d.data_ptr<float>()),
                ray_transformations.data_ptr<float>(), colors.data_ptr<float>(),
                reinterpret_cast<vec3<float> *>(normals.data_ptr<float>()), opacities.data_ptr<float>(),
                backgrounds.has_value() ? backgrounds.value().data_ptr<float>()
                                        : nullptr,
                image_width, image_height, tile_size, tile_width, tile_height,
                tile_offsets.data_ptr<int32_t>(), flatten_ids.data_ptr<int32_t>(),
                render_alphas.data_ptr<float>(), 
                last_ids.data_ptr<int32_t>(),
                v_render_colors.data_ptr<float>(), v_render_alphas.data_ptr<float>(),
                v_render_normals.data_ptr<float>(),
                absgrad ? reinterpret_cast<vec2<float> *>(v_means2d_abs.data_ptr<float>()) : nullptr,
                reinterpret_cast<vec2<float> *>(v_means2d.data_ptr<float>()),
                v_ray_transformations.data_ptr<float>(), v_colors.data_ptr<float>(),
                v_opacities.data_ptr<float>(), v_normals.data_ptr<float>()
            );
            break;
        case 32:
            rasterize_to_pixels_bwd_2dgs_kernel<32, float><<<blocks, threads, shared_mem, stream>>>(
                C, N, n_isects, packed, reinterpret_cast<vec2<float> *>(means2d.data_ptr<float>()),
                ray_transformations.data_ptr<float>(), colors.data_ptr<float>(),
                reinterpret_cast<vec3<float> *>(normals.data_ptr<float>()), opacities.data_ptr<float>(),
                backgrounds.has_value() ? backgrounds.value().data_ptr<float>()
                                        : nullptr,
                image_width, image_height, tile_size, tile_width, tile_height,
                tile_offsets.data_ptr<int32_t>(), flatten_ids.data_ptr<int32_t>(),
                render_alphas.data_ptr<float>(), 
                last_ids.data_ptr<int32_t>(),
                v_render_colors.data_ptr<float>(), v_render_alphas.data_ptr<float>(),
                v_render_normals.data_ptr<float>(),
                absgrad ? reinterpret_cast<vec2<float> *>(v_means2d_abs.data_ptr<float>()) : nullptr,
                reinterpret_cast<vec2<float> *>(v_means2d.data_ptr<float>()),
                v_ray_transformations.data_ptr<float>(), v_colors.data_ptr<float>(),
                v_opacities.data_ptr<float>(), v_normals.data_ptr<float>()
            );
            break;
        default:
            AT_ERROR("Unsupported number of channels: ", COLOR_DIM);
        }
    }

    return std::make_tuple(v_means2d_abs, v_means2d, v_ray_transformations, v_colors, v_opacities, v_normals);
}


// template <uint32_t COLOR_DIM>
// __global__ void rasterize_to_pixels_bwd_2dgs_kernel(
//     const uint32_t C, const uint32_t N, const uint32_t n_isects, const bool packed,
//     // fwd inputs
//     const float2 *__restrict__ means2d,
//     const float *__restrict__ ray_transformations,
//     const float *__restrict__ colors,
//     const float3 *__restrict__ normals,
//     const float *__restrict__ opacities,
//     const float *__restrict__ backgrounds,
//     const uint32_t image_width, const uint32_t image_height, const uint32_t tile_size,
//     const uint32_t tile_width, const uint32_t tile_height,
//     const int32_t *__restrict__ tile_offsets,
//     const int32_t *__restrict__ flatten_ids,
//     // fwd outputs
//     const float *__restrict__ render_alphas,
//     const int32_t *__restrict__ last_ids,
//     // grad outputs
//     const float *__restrict__ v_render_colors,
//     const float *__restrict__ v_render_alphas,
//     const float *__restrict__ v_render_normals,
//     // grad inputs
//     float2 *__restrict__ v_means2d_abs,
//     float2 *__restrict__ v_means2d,
//     float *__restrict__ v_ray_transformations,
//     float *__restrict__ v_colors,
//     float *__restrict__ v_opacities,
//     float *__restrict__ v_normal3d
// ) {
//     auto block = cg::this_thread_block();
//     uint32_t camera_id = block.group_index().x;
//     uint32_t tile_id = block.group_index().y * tile_width + block.group_index().z;
//     uint32_t i = block.group_index().y * tile_size + block.thread_index().y;
//     uint32_t j = block.group_index().z * tile_size + block.thread_index().x;

//     tile_offsets += camera_id * tile_height * tile_width;
//     render_alphas += camera_id * image_height * image_width;
//     last_ids += camera_id * image_height * image_width;
//     v_render_colors += camera_id * image_height * image_width * COLOR_DIM;
//     v_render_alphas += camera_id * image_height * image_width;
//     v_render_normals += camera_id * image_height * image_width * 3;
//     if (backgrounds != nullptr) {
//         backgrounds += camera_id * COLOR_DIM;
//     }


//     const float px = (float)j + 0.5f;
//     const float py = (float)i + 0.5f;
//     // clamp this value to the last pixel
//     const int32_t pix_id = min(i * image_width + j, image_width * image_height - 1);

//     // keep not rasterizing threads around for reading data
//     bool inside = (i < image_height && j < image_width);

//     // have all threads in tile process the same gaussians in batches
//     // first collect gaussians between range.x and range.y in batches
//     // which gaussians to look through in this tile
//     int32_t range_start = tile_offsets[tile_id];
//     int32_t range_end = 
//         (camera_id == C - 1) && (tile_id == tile_width * tile_height - 1)
//             ? n_isects
//             : tile_offsets[tile_id + 1];
//     const uint32_t block_size = block.size();
//     const uint32_t num_batches = 
//         (range_end - range_start + block_size - 1) / block_size;
    
//     __shared__ int32_t id_batch[MAX_BLOCK_SIZE];
//     __shared__ float3 xy_opacity_batch[MAX_BLOCK_SIZE];
//     __shared__ float3 u_transform_batch[MAX_BLOCK_SIZE];
//     __shared__ float3 v_transform_batch[MAX_BLOCK_SIZE];
//     __shared__ float3 w_transform_batch[MAX_BLOCK_SIZE];
//     __shared__ float rgbs_batch[MAX_BLOCK_SIZE * COLOR_DIM];
//     __shared__ float normals_batch[MAX_BLOCK_SIZE * 3];

//     // this is the T AFTER the last gaussian in this pixel
//     float T_final = 1.0f - render_alphas[pix_id];
//     float T = T_final;
//     // the contribution from gaussians behind the current one
//     float buffer[COLOR_DIM] = {0.f};
//     float buffer_normals[3] = {0.f};
//     // index of last gaussian to contribute to this pixel
//     const int32_t bin_final = inside ? last_ids[pix_id] : 0;

//     // df/d_out for this pixel
//     float v_render_c[COLOR_DIM];
//     PRAGMA_UNROLL
//     for (uint32_t k = 0; k < COLOR_DIM; ++k) {
//         v_render_c[k] = v_render_colors[pix_id * COLOR_DIM + k];
//     }
//     const float v_render_a = v_render_alphas[pix_id];
//     float v_render_n[3];
//     PRAGMA_UNROLL
//     for (uint32_t k = 0; k < 3; ++k) {
//         v_render_n[k] = v_render_normals[pix_id * 3 + k];
//     }



//     // collect and process batches of gaussians
//     // each thread loads one gaussian at a time before rasterizing
//     const uint32_t tr = block.thread_rank();
//     cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
//     const int32_t warp_bin_final = cg::reduce(warp, bin_final, cg::greater<int>());
//     for (uint32_t b = 0; b < num_batches; ++b) {
//         // resync all threads before writing next batch of shared mem
//         block.sync();

//         // each thread fetch 1 gaussian from back to front 
//         // 0 index will be furthest back in batch
//         // index of gaussian to load
//         // batch end is the index of the last gaussian in the batch
//         // These values can be negative so must be int32 instead of uint32
//         const int32_t batch_end = range_end - 1 - block_size * b;
//         const int32_t batch_size = min(block_size, batch_end + 1 - range_start);
//         const int32_t idx = batch_end - tr;
//         if (idx >= range_start) {
//             int32_t g = flatten_ids[idx]; // flatten index in [C * N] or [nnz]
//             id_batch[tr] = g;
//             const float2 xy = means2d[g];
//             const float opac = opacities[g];
//             xy_opacity_batch[tr] = {xy.x, xy.y, opac};
//             u_transform_batch[tr] = {ray_transformations[g * 9 + 0], ray_transformations[g * 9 + 1], ray_transformations[g * 9 + 2]};
//             v_transform_batch[tr] = {ray_transformations[g * 9 + 3], ray_transformations[g * 9 + 4], ray_transformations[g * 9 + 5]};
//             w_transform_batch[tr] = {ray_transformations[g * 9 + 6], ray_transformations[g * 9 + 7], ray_transformations[g * 9 + 8]};

//             const float3 normal = normals[g];
//             normals_batch[tr * 3] = normal.x;
//             normals_batch[tr * 3 + 1] = normal.y;
//             normals_batch[tr * 3 + 2] = normal.z;

//             PRAGMA_UNROLL
//             for (uint32_t k = 0; k < COLOR_DIM; ++k) {
//                 rgbs_batch[tr * COLOR_DIM + k] = colors[g * COLOR_DIM + k];
//             }

//         }

//         // wait for other threads to collect the gaussians in batch
//         block.sync();
//         // process gaussians in the current batch for this pixel
//         // 0 index is the furthest back gaussian in the batch
//         for (uint32_t t = max(0, batch_end - warp_bin_final); t < batch_size; ++t) {
//             bool valid = inside;
//             if (batch_end - t > bin_final) {
//                 valid = 0;
//             }
//             float alpha;
//             float opac;
//             float vis; 
//             float gauss_weight_3d;
//             float gauss_weight_2d;
//             float gauss_weight;
//             float2 s;
//             float2 d;
//             float3 h_u;
//             float3 h_v;
//             float3 intersect;
//             float3 w_transform;
//             if (valid) { 
//                 float3 xy_opac = xy_opacity_batch[t];
//                 opac = xy_opac.z;
//                 const float3 u_transform = u_transform_batch[t];
//                 const float3 v_transform = v_transform_batch[t];
//                 w_transform = w_transform_batch[t];

//                 h_u = px * w_transform - u_transform;
//                 h_v = py * w_transform - v_transform;
                

//                 // cross product of two planes is a line
//                 intersect = cross_product(h_u, h_v);

//                 // No intersection
//                 if (intersect.z == 0.0) valid = false;
//                 s = {intersect.x / intersect.z, intersect.y / intersect.z};

//                 gauss_weight_3d = f2_norm2(s);
//                 d = {xy_opac.x - px, xy_opac.y - py};
//                 gauss_weight_2d = FilterInvSquare * f2_norm2(d);

//                 gauss_weight = min(gauss_weight_3d, gauss_weight_2d);
//                 const float sigma = 0.5f * gauss_weight;
//                 vis = __expf(-sigma);
//                 alpha = min(0.999f, opac * vis);
//                 if (sigma < 0.f || alpha < 1.f / 255.f) {
//                     valid = false;
//                 }
//             }


//             // if all threads are inactive in this warp, skip this loop
//             if (!warp.any(valid)) {
//                 continue;
//             }
//             float v_rgb_local[COLOR_DIM] = {0.f};
//             float3 v_u_transform_local = {0.f, 0.f, 0.f};
//             float3 v_v_transform_local = {0.f, 0.f, 0.f};
//             float3 v_w_transform_local = {0.f, 0.f, 0.f};
//             float2 v_xy_local = {0.f, 0.f};
//             float2 v_xy_abs_local = {0.f, 0.f};
//             float v_opacity_local = 0.f;
//             float2 v_densification_local = {0.f, 0.f};
//             float v_normal_local[3] = {0.f};
//             // initialize everything to 0, only set if the lane is valid
//             if (valid) {
//                 // compute the current T for this gaussian
//                 float ra = 1.0f / (1.0f - alpha);
//                 T *= ra;
//                 // update v_rgb for this gaussian
//                 const float fac = alpha * T;
//                 PRAGMA_UNROLL
//                 for (uint32_t k = 0; k < COLOR_DIM; ++k) {
//                     v_rgb_local[k] = fac * v_render_c[k];
//                 }
//                 // contribution from this pixel
//                 float v_alpha = 0.f;
//                 for (uint32_t k = 0; k < COLOR_DIM; ++k) {
//                     v_alpha += (rgbs_batch[t * COLOR_DIM + k] * T - buffer[k] * ra) *
//                                 v_render_c[k];
//                 }

//                 // update v_normal for this gaussian
//                 // TODO (WZ): derive the computational graph to see if the gradient flow
//                 // is correct or not.
//                 PRAGMA_UNROLL
//                 for (uint32_t k = 0; k < 3; ++k) {
//                     v_normal_local[k] = fac * v_render_n[k];
//                 }

//                 for (uint32_t k = 0; k < 3; ++k) {
//                     v_alpha += (normals_batch[t * 3 + k] * T - buffer_normals[k] * ra) *
//                                 v_render_n[k];
//                 }

//                 v_alpha += T_final * ra * v_render_a;

//                 // contribution from background pixel
//                 if (backgrounds != nullptr) {
//                     float accum = 0.f;
//                     PRAGMA_UNROLL
//                     for (uint32_t k = 0; k < COLOR_DIM; ++k) {
//                         accum += backgrounds[k] * v_render_c[k];
//                     }
//                     v_alpha += -T_final * ra * accum;
//                 }


//                 //====== 2DGS ======//
//                 if (opac * vis <= 0.999f) {
//                     const float v_G = opac * v_alpha;
//                     float v_depth = 0.f;
//                     if (gauss_weight_3d <= gauss_weight_2d) {
//                         const float2 v_s = {
//                             v_G * -vis * s.x + v_depth * w_transform.x,
//                             v_G * -vis * s.y + v_depth * w_transform.y
//                         };
//                         const float3 v_z_w_transform = {s.x, s.y, 1.0};
//                         const float v_sx_pz = v_s.x / intersect.z;
//                         const float v_sy_pz = v_s.y / intersect.z;
//                         const float3 v_intersect = {v_sx_pz, v_sy_pz, -(v_sx_pz * s.x + v_sy_pz * s.y)};
//                         const float3 v_h_u = cross_product(h_v, v_intersect);
//                         const float3 v_h_v = cross_product(v_intersect, h_u);
                        
//                         v_u_transform_local = {-v_h_u.x, -v_h_u.y, -v_h_u.z};
//                         v_v_transform_local = {-v_h_v.x, -v_h_v.y, -v_h_v.z};
//                         v_w_transform_local = {
//                             px * v_h_u.x + py * v_h_v.x + v_depth * v_z_w_transform.x,
//                             px * v_h_u.y + py * v_h_v.y + v_depth * v_z_w_transform.y,
//                             px * v_h_u.z + py * v_h_v.z + v_depth * v_z_w_transform.z
//                         };
                        
//                     } else {
//                         const float v_G_ddelx = -vis * FilterInvSquare * d.x;
//                         const float v_G_ddely = -vis * FilterInvSquare * d.y;
//                         v_xy_local = {v_G * v_G_ddelx, v_G * v_G_ddely};
//                     }

//                     v_opacity_local = vis * v_alpha;
//                 }

//                 PRAGMA_UNROLL
//                 for (uint32_t k = 0; k < COLOR_DIM; ++k) {
//                     buffer[k] += rgbs_batch[t * COLOR_DIM + k] * fac;
//                 }

//                 PRAGMA_UNROLL
//                 for (uint32_t k = 0; k < 3; ++k) {
//                     buffer_normals[k] += normals_batch[t * 3 + k] * fac;
//                 }
                
//                 float depth = w_transform.z;
//                 v_densification_local.x = v_u_transform_local.z * depth;
//                 v_densification_local.y = v_v_transform_local.z * depth;
//             }
//             warpSum<COLOR_DIM, float>(v_rgb_local, warp);
//             warpSum<3, float>(v_normal_local, warp);
//             warpSum(v_xy_local, warp);
//             if (v_means2d_abs != nullptr) {
//                 warpSum(v_xy_abs_local, warp);
//             }
//             warpSum(v_opacity_local, warp);
//             warpSum(v_u_transform_local, warp);
//             warpSum(v_v_transform_local, warp);
//             warpSum(v_w_transform_local, warp);
//             warpSum(v_densification_local, warp);
//             if (warp.thread_rank() == 0) {
//                 int32_t g = id_batch[t]; // flatten index in [C * N] or [nnz]
//                 float *v_rgb_ptr = (float *)(v_colors) + COLOR_DIM * g;
//                 PRAGMA_UNROLL
//                 for (uint32_t k = 0; k < COLOR_DIM; ++k) {
//                     atomicAdd(v_rgb_ptr + k, v_rgb_local[k]);
//                 }

//                 //====== 2DGS ======//
//                 // if (gauss_weight_3d <= gauss_weight_2d) {
//                 float *v_ray_transformation_ptr = (float *)(v_ray_transformations) + 9 * g;
//                 atomicAdd(v_ray_transformation_ptr, v_u_transform_local.x);
//                 atomicAdd(v_ray_transformation_ptr + 1, v_u_transform_local.y);
//                 atomicAdd(v_ray_transformation_ptr + 2, v_u_transform_local.z);
//                 atomicAdd(v_ray_transformation_ptr + 3, v_v_transform_local.x);
//                 atomicAdd(v_ray_transformation_ptr + 4, v_v_transform_local.y);
//                 atomicAdd(v_ray_transformation_ptr + 5, v_v_transform_local.z);
//                 atomicAdd(v_ray_transformation_ptr + 6, v_w_transform_local.x);
//                 atomicAdd(v_ray_transformation_ptr + 7, v_w_transform_local.y);
//                 atomicAdd(v_ray_transformation_ptr + 8, v_w_transform_local.z);
//                 // } else {
//                 // printf("%.2f, %.2f \n", gauss_weight_3d, gauss_weight_2d);
//                 float *v_xy_ptr = (float *)(v_means2d) + 2 * g;
//                 atomicAdd(v_xy_ptr, v_xy_local.x);
//                 atomicAdd(v_xy_ptr + 1, v_xy_local.y);

//                 if (v_means2d_abs != nullptr) {
//                     float *v_xy_abs_ptr = (float *)(v_means2d_abs) + 2 * g;
//                     atomicAdd(v_xy_abs_ptr, v_xy_abs_local.x);
//                     atomicAdd(v_xy_abs_ptr + 1, v_xy_abs_local.y);
//                 }
//                 // }


//                 float *v_normal_ptr = (float *)(v_normal3d) + 3 * g;
//                 PRAGMA_UNROLL
//                 for (uint32_t k = 0; k < 3; ++k) {
//                     atomicAdd(v_normal_ptr + k, v_normal_local[k]);
//                 }
                
//                 atomicAdd(v_opacities + g, v_opacity_local);
//             }
//         }
//     }
// }

// std::tuple<torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor>
// rasterize_to_pixels_bwd_2dgs_tensor(
//     // Gaussian parameters
//     const torch::Tensor &means2d,
//     const torch::Tensor &ray_transformations,
//     const torch::Tensor &colors,
//     const torch::Tensor &opacities,
//     const torch::Tensor &normals,
//     const at::optional<torch::Tensor> &backgrounds,
//     // image size
//     const uint32_t image_width, const uint32_t image_height, const uint32_t tile_size,
//     // intersection
//     const torch::Tensor &tile_offsets,
//     const torch::Tensor &flatten_ids,
//     // forward outputs
//     const torch::Tensor &render_alphas,
//     const torch::Tensor &last_ids,
//     // gradients of outputs
//     const torch::Tensor &v_render_colors,
//     const torch::Tensor &v_render_alphas,
//     const torch::Tensor &v_render_normals,
//     // options
//     bool absgrad
// ) {
//     DEVICE_GUARD(means2d);
//     CHECK_INPUT(means2d);
//     CHECK_INPUT(ray_transformations);
//     CHECK_INPUT(colors);
//     CHECK_INPUT(opacities);
//     CHECK_INPUT(normals);
//     CHECK_INPUT(tile_offsets);
//     CHECK_INPUT(flatten_ids);
//     CHECK_INPUT(render_alphas);
//     CHECK_INPUT(last_ids);
//     CHECK_INPUT(v_render_colors);
//     CHECK_INPUT(v_render_alphas);
//     CHECK_INPUT(v_render_normals);
//     if (backgrounds.has_value()) {
//         CHECK_INPUT(backgrounds.value());
//     }

//     bool packed = means2d.dim() == 2;

//     uint32_t C = tile_offsets.size(0);         // number of cameras
//     uint32_t N = packed ? 0 : means2d.size(1); // number of gaussians
//     uint32_t n_isects = flatten_ids.size(0);
//     uint32_t COLOR_DIM = colors.size(-1);
//     uint32_t tile_height = tile_offsets.size(1);
//     uint32_t tile_width = tile_offsets.size(2);

//     // Each block covers a tile on the image. In total there are
//     // C * tile_height * tile_width blocks.
//     dim3 threads = {tile_size, tile_size, 1};
//     dim3 blocks = {C, tile_height, tile_width};

//     torch::Tensor v_means2d = torch::zeros_like(means2d);
//     torch::Tensor v_ray_transformations = torch::zeros_like(ray_transformations);
//     torch::Tensor v_colors = torch::zeros_like(colors);
//     torch::Tensor v_opacities = torch::zeros_like(opacities);
//     torch::Tensor v_means2d_abs;
//     if (absgrad) {
//         v_means2d_abs = torch::zeros_like(means2d);
//     }
//     torch::Tensor v_normals = torch::zeros_like(normals);
//     if (n_isects) {
//         at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();
//         switch (COLOR_DIM) {
//         case 1:
//             rasterize_to_pixels_bwd_2dgs_kernel<1><<<blocks, threads, 0, stream>>>(
//                 C, N, n_isects, packed, (float2 *)means2d.data_ptr<float>(),
//                 ray_transformations.data_ptr<float>(), colors.data_ptr<float>(),
//                 (float3 *)normals.data_ptr<float>(), opacities.data_ptr<float>(),
//                 backgrounds.has_value() ? backgrounds.value().data_ptr<float>()
//                                         : nullptr,
//                 image_width, image_height, tile_size, tile_width, tile_height,
//                 tile_offsets.data_ptr<int32_t>(), flatten_ids.data_ptr<int32_t>(),
//                 render_alphas.data_ptr<float>(), 
//                 last_ids.data_ptr<int32_t>(),
//                 v_render_colors.data_ptr<float>(), v_render_alphas.data_ptr<float>(),
//                 v_render_normals.data_ptr<float>(),
//                 absgrad ? (float2 *)v_means2d_abs.data_ptr<float>() : nullptr,
//                 (float2 *)v_means2d.data_ptr<float>(),
//                 v_ray_transformations.data_ptr<float>(), v_colors.data_ptr<float>(),
//                 v_opacities.data_ptr<float>(), v_normals.data_ptr<float>()
//             );
//             break;
//         case 2:
//             rasterize_to_pixels_bwd_2dgs_kernel<2><<<blocks, threads, 0, stream>>>(
//                 C, N, n_isects, packed, (float2 *)means2d.data_ptr<float>(),
//                 ray_transformations.data_ptr<float>(), colors.data_ptr<float>(),
//                 (float3 *)normals.data_ptr<float>(), opacities.data_ptr<float>(),
//                 backgrounds.has_value() ? backgrounds.value().data_ptr<float>()
//                                         : nullptr,
//                 image_width, image_height, tile_size, tile_width, tile_height,
//                 tile_offsets.data_ptr<int32_t>(), flatten_ids.data_ptr<int32_t>(),
//                 render_alphas.data_ptr<float>(), 
//                 last_ids.data_ptr<int32_t>(),
//                 v_render_colors.data_ptr<float>(), v_render_alphas.data_ptr<float>(),
//                 v_render_normals.data_ptr<float>(),
//                 absgrad ? (float2 *)v_means2d_abs.data_ptr<float>() : nullptr,
//                 (float2 *)v_means2d.data_ptr<float>(),
//                 v_ray_transformations.data_ptr<float>(), v_colors.data_ptr<float>(),
//                 v_opacities.data_ptr<float>(), v_normals.data_ptr<float>()
//             );
//             break;
//         case 3:
//             rasterize_to_pixels_bwd_2dgs_kernel<3><<<blocks, threads, 0, stream>>>(
//                 C, N, n_isects, packed, (float2 *)means2d.data_ptr<float>(),
//                 ray_transformations.data_ptr<float>(), colors.data_ptr<float>(),
//                 (float3 *)normals.data_ptr<float>(), opacities.data_ptr<float>(),
//                 backgrounds.has_value() ? backgrounds.value().data_ptr<float>()
//                                         : nullptr,
//                 image_width, image_height, tile_size, tile_width, tile_height,
//                 tile_offsets.data_ptr<int32_t>(), flatten_ids.data_ptr<int32_t>(),
//                 render_alphas.data_ptr<float>(), 
//                 last_ids.data_ptr<int32_t>(),
//                 v_render_colors.data_ptr<float>(), v_render_alphas.data_ptr<float>(),
//                 v_render_normals.data_ptr<float>(),
//                 absgrad ? (float2 *)v_means2d_abs.data_ptr<float>() : nullptr,
//                 (float2 *)v_means2d.data_ptr<float>(),
//                 v_ray_transformations.data_ptr<float>(), v_colors.data_ptr<float>(),
//                 v_opacities.data_ptr<float>(), v_normals.data_ptr<float>()
//             );
//             break;     
//         case 4:
//             rasterize_to_pixels_bwd_2dgs_kernel<4><<<blocks, threads, 0, stream>>>(
//                 C, N, n_isects, packed, (float2 *)means2d.data_ptr<float>(),
//                 ray_transformations.data_ptr<float>(), colors.data_ptr<float>(),
//                 (float3 *)normals.data_ptr<float>(), opacities.data_ptr<float>(),
//                 backgrounds.has_value() ? backgrounds.value().data_ptr<float>()
//                                         : nullptr,
//                 image_width, image_height, tile_size, tile_width, tile_height,
//                 tile_offsets.data_ptr<int32_t>(), flatten_ids.data_ptr<int32_t>(),
//                 render_alphas.data_ptr<float>(), 
//                 last_ids.data_ptr<int32_t>(),
//                 v_render_colors.data_ptr<float>(), v_render_alphas.data_ptr<float>(),
//                 v_render_normals.data_ptr<float>(),
//                 absgrad ? (float2 *)v_means2d_abs.data_ptr<float>() : nullptr,
//                 (float2 *)v_means2d.data_ptr<float>(),
//                 v_ray_transformations.data_ptr<float>(), v_colors.data_ptr<float>(),
//                 v_opacities.data_ptr<float>(), v_normals.data_ptr<float>()
//             );
//             // CUDA_CHECK_ERROR;
//             // CUDA_SAFE_CALL(hipStreamSynchronize(stream.stream()));
//             break;
//         case 8:
//             rasterize_to_pixels_bwd_2dgs_kernel<8><<<blocks, threads, 0, stream>>>(
//                 C, N, n_isects, packed, (float2 *)means2d.data_ptr<float>(),
//                 ray_transformations.data_ptr<float>(), colors.data_ptr<float>(),
//                 (float3 *)normals.data_ptr<float>(), opacities.data_ptr<float>(),
//                 backgrounds.has_value() ? backgrounds.value().data_ptr<float>()
//                                         : nullptr,
//                 image_width, image_height, tile_size, tile_width, tile_height,
//                 tile_offsets.data_ptr<int32_t>(), flatten_ids.data_ptr<int32_t>(),
//                 render_alphas.data_ptr<float>(), 
//                 last_ids.data_ptr<int32_t>(),
//                 v_render_colors.data_ptr<float>(), v_render_alphas.data_ptr<float>(),
//                 v_render_normals.data_ptr<float>(),
//                 absgrad ? (float2 *)v_means2d_abs.data_ptr<float>() : nullptr,
//                 (float2 *)v_means2d.data_ptr<float>(),
//                 v_ray_transformations.data_ptr<float>(), v_colors.data_ptr<float>(),
//                 v_opacities.data_ptr<float>(), v_normals.data_ptr<float>()
//             );
//             break;
//         case 16:
//             rasterize_to_pixels_bwd_2dgs_kernel<16><<<blocks, threads, 0, stream>>>(
//                 C, N, n_isects, packed, (float2 *)means2d.data_ptr<float>(),
//                 ray_transformations.data_ptr<float>(), colors.data_ptr<float>(),
//                 (float3 *)normals.data_ptr<float>(), opacities.data_ptr<float>(),
//                 backgrounds.has_value() ? backgrounds.value().data_ptr<float>()
//                                         : nullptr,
//                 image_width, image_height, tile_size, tile_width, tile_height,
//                 tile_offsets.data_ptr<int32_t>(), flatten_ids.data_ptr<int32_t>(),
//                 render_alphas.data_ptr<float>(), 
//                 last_ids.data_ptr<int32_t>(),
//                 v_render_colors.data_ptr<float>(), v_render_alphas.data_ptr<float>(),
//                 v_render_normals.data_ptr<float>(),
//                 absgrad ? (float2 *)v_means2d_abs.data_ptr<float>() : nullptr,
//                 (float2 *)v_means2d.data_ptr<float>(),
//                 v_ray_transformations.data_ptr<float>(), v_colors.data_ptr<float>(),
//                 v_opacities.data_ptr<float>(), v_normals.data_ptr<float>()
//             );
//             break;
//         case 32:
//             rasterize_to_pixels_bwd_2dgs_kernel<32><<<blocks, threads, 0, stream>>>(
//                 C, N, n_isects, packed, (float2 *)means2d.data_ptr<float>(),
//                 ray_transformations.data_ptr<float>(), colors.data_ptr<float>(),
//                 (float3 *)normals.data_ptr<float>(), opacities.data_ptr<float>(),
//                 backgrounds.has_value() ? backgrounds.value().data_ptr<float>()
//                                         : nullptr,
//                 image_width, image_height, tile_size, tile_width, tile_height,
//                 tile_offsets.data_ptr<int32_t>(), flatten_ids.data_ptr<int32_t>(),
//                 render_alphas.data_ptr<float>(), 
//                 last_ids.data_ptr<int32_t>(),
//                 v_render_colors.data_ptr<float>(), v_render_alphas.data_ptr<float>(),
//                 v_render_normals.data_ptr<float>(),
//                 absgrad ? (float2 *)v_means2d_abs.data_ptr<float>() : nullptr,
//                 (float2 *)v_means2d.data_ptr<float>(),
//                 v_ray_transformations.data_ptr<float>(), v_colors.data_ptr<float>(),
//                 v_opacities.data_ptr<float>(), v_normals.data_ptr<float>()
//             );
//             break;
//         default:
//             AT_ERROR("Unsupported number of channels: ", COLOR_DIM);
//         }
//     }

//     return std::make_tuple(v_means2d_abs, v_means2d, v_ray_transformations, v_colors, v_opacities, v_normals);
// }